#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <chrono>

using namespace std;


__device__ unsigned int makeInt(const unsigned short a, const unsigned short b) {
	return (a << 16) | (b & 0xFFFF);
}

template<unsigned index>
__device__ unsigned short extract(const unsigned int a) {
	if (index == 0) {
		return (unsigned short)(a >> 16);
	}
	else {
		return (unsigned short)(a & 0xffff);
	}
}

void print(char* m, int height, int width) {
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			cout << (unsigned int)m[i * width + j] << " ";
		}
		cout << "\n";
	}
}

__global__ void flatMatrixKernel(const char* src, char* dst, int height, int width) {

	const uint2 offset = make_uint2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	if ((offset.x / 2) % 2 == 0) {
		if (offset.x == 0 || offset.x == 1) {

			dst[offset.y * width + offset.x] = src[offset.y * width + offset.x];
		}
		else {
			dst[offset.y * width + offset.x - (offset.x / 2)] = src[offset.y * width + offset.x];
		}
	}
	else {
		dst[(offset.y * 2 + 1) * width / 2 + offset.x - (offset.x / 2 + 1)] = src[offset.y * width + offset.x];
	}
}


__global__ void flatMatrixKernelShared(uint32_t* src, uint32_t* dst, int src_pitch, int dst_pitch, int height, int width) {

	__shared__ uint32_t shared[32 * 32];

	const uint2 offset = make_uint2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	int src_width = src_pitch / 4;
	int dst_width = dst_pitch / 4;

	if (offset.x < width && offset.y < height) {
		shared[threadIdx.y * blockDim.x + threadIdx.x] = src[offset.y * src_width + offset.x];
	}
	__syncthreads();

	unsigned short firstPart = extract<1>(shared[threadIdx.y * blockDim.x + threadIdx.x]);
	unsigned short secondPart = extract<0>(shared[threadIdx.y * blockDim.x + threadIdx.x]);

	if (offset.x < width && offset.y < height) {
		if (offset.x % 2 == 0) {
			unsigned short firstPartResult = extract<0>(
				dst[offset.y * 2 * dst_width + (offset.x / 2)]);
			unsigned short firstPartResultB = extract<0>(
				dst[(offset.y * 2 + 1) * dst_width + (offset.x / 2)]);
			dst[offset.y * 2 * dst_width + (offset.x / 2)] = makeInt(firstPartResult, firstPart);
			dst[(offset.y * 2 + 1) * dst_width + (offset.x / 2)] = makeInt(firstPartResultB, secondPart);
		}
		else {
			unsigned short firstPartResult = extract<1>(
				dst[offset.y * 2 * dst_width + (offset.x / 2)]);
			unsigned short firstPartResultB = extract<1>(
				dst[(offset.y * 2 + 1) * dst_width + (offset.x / 2)]);
			dst[offset.y * 2 * dst_width + (offset.x / 2)] = makeInt(firstPart, firstPart);
			dst[(offset.y * 2 + 1) * dst_width + (offset.x / 2)] = makeInt(secondPart, firstPartResultB);
		}
	}
}

void initialize(char* src, int height, int width) {
	for (int i = 0; i < height * width; i++) {
		src[i] = i;
	}
}

bool compare(char* firstMatrix, char* secondMatrix, int size) {
	for (int i = 0; i < size; i++) {
		if (firstMatrix[i] != secondMatrix[i]) {
			return false;
		}
	}
	return true;
}

void flatMatrix(char* src, char* dst, int width, int height) {
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			if ((j / 2) % 2 == 0) {
				if (j == 0 || j == 1) {
					dst[i * width + j] = src[i * width + j];
				}
				else {
					dst[i * width + j - (j / 2)] = src[i * width + j];
				}
			}
			else {
				dst[(i * 2 + 1) * width / 2 + j - (j / 2 + 1)] = src[i * width + j];
			}
		}
	}
}

float flatMatrixBasic(char* src, char* dst, int height, int width) {
	
	const int size = height * width;
	char* d_src = nullptr;
	char* d_dst = nullptr;
	hipStream_t stream = 0;
	hipEvent_t eventStart, eventStop;
	hipEventCreate(&eventStart);
	hipEventCreate(&eventStop);

	if (hipMalloc((void**)&d_src, size * sizeof(char)) != hipSuccess) {
		fprintf(stderr, "Error flatMatrixBasic() 1");
		hipFree(d_src);
		hipFree(d_dst);
		return -1;
	}

	if (hipMalloc((void**)&d_dst, size * sizeof(char)) != hipSuccess) {
		fprintf(stderr, "Error flatMatrixBasic() 2");
		hipFree(d_src);
		hipFree(d_dst);
		return -1;
	}

	if (hipMemcpy(d_src, src, size * sizeof(char), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error flatMatrixBasic() 3");
		hipFree(d_src);
		hipFree(d_dst);
		return -1;
	}

	dim3 blockDimension(32, 32);
	dim3 gridDimension(width / blockDimension.x, height / blockDimension.y);

	hipEventRecord(eventStart, stream);
	flatMatrixKernel << < gridDimension, blockDimension >> > (d_src, d_dst, height, width);

	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "Error flatMatrixBasic() 4");
		hipFree(d_src);
		hipFree(d_dst);
		return -1;
	}

	if (hipMemcpy(dst, d_dst, size * sizeof(char), hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "Error flatMatrixBasic() 5");
		hipFree(d_src);
		hipFree(d_dst);
		return -1;
	}

	hipEventRecord(eventStop, stream);
	hipEventSynchronize(eventStop);
	float occupiedTime;
	hipEventElapsedTime(&occupiedTime, eventStart, eventStop);

	hipFree(d_src);
	hipFree(d_dst);

	return occupiedTime;
}

float flatMatrixShared(char* src, char* dst, int height, int width) {

	const int size = height * width;
	uint32_t* d_src = nullptr;
	uint32_t* d_dst = nullptr;
	hipStream_t stream = 0;
	size_t src_pitch = 0;
	size_t dst_pitch = 0;
	hipEvent_t eventStart, eventStop;
	hipEventCreate(&eventStart);
	hipEventCreate(&eventStop);

	if (hipMallocPitch((void**)&d_src, &src_pitch, width * sizeof(char), height) != hipSuccess) {
		fprintf(stderr, "Error flatMatrixShared() 1");
		hipFree(d_src);
		hipFree(d_dst);
		return -1;
	}

	if (hipMallocPitch((void**)&d_dst, &dst_pitch, width * sizeof(char) / 2, height * 2) != hipSuccess) {
		fprintf(stderr, "Error flatMatrixShared() 2");
		hipFree(d_src);
		hipFree(d_dst);
		return -1;
	}

	if (hipMemcpy2D(d_src, src_pitch, src, width * sizeof(char), width * sizeof(char), height,
		hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Error flatMatrixShared() 3");
		hipFree(d_src);
		hipFree(d_dst);
		return -1;
	}

	dim3 blockDimension(32, 32);
	dim3 gridDimension((width + blockDimension.x - 1) / blockDimension.x,
		(height + blockDimension.y - 1) / blockDimension.y);
	hipEventRecord(eventStart, stream);
	flatMatrixKernelShared << < gridDimension, blockDimension >> > (d_src, d_dst, src_pitch, dst_pitch, height, width / 4);

	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "Error flatMatrixShared() 4");
		hipFree(d_src);
		hipFree(d_dst);
		return -1;
	}

	if (hipMemcpy2D(dst, width * sizeof(char) / 2, d_dst, dst_pitch, width * sizeof(char) / 2, height * 2,
		hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "Error flatMatrixShared() 5");
		hipFree(d_src);
		hipFree(d_dst);
		return -1;
	}

	hipEventRecord(eventStop, stream);
	hipEventSynchronize(eventStop);

	float occupiedTime;
	hipEventElapsedTime(&occupiedTime, eventStart, eventStop);

	hipFree(d_src);
	hipFree(d_dst);

	return occupiedTime;
}

int main() {

	int n = 7;
	int m = 3;
	int n_new = n;
	if (n % 4 != 0) {
		for (n_new = n;; n_new++) {
			if (n_new % 4 == 0) {
				break;
			}
		}
	}
	cout << n_new - n;
	char* h_src = (char*)malloc(n_new * m * sizeof(char));
	char* d_dst = (char*)malloc(n_new * m * sizeof(char));
	char* d_dst_for_basic = (char*)malloc(n_new * m * sizeof(char));
	char* d_dst_for_not_full = (char*)malloc(n_new * m * sizeof(char));
	char* d_dst_for_shared = (char*)malloc(n_new * m * sizeof(char));

	initialize(h_src, m, n_new);

	int od = (n_new - n) * m;
	cout << od;
	for (int i = 0; i < od; i++) {
		h_src[m * n_new - i - 1] = 0;
	}
	auto t0 = std::chrono::high_resolution_clock::now();
	flatMatrix(h_src, d_dst, n, m);
	auto t1 = std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time = std::chrono::duration_cast<std::chrono::duration<double>>(t1 - t0);

	print(h_src, m, n_new);

	cout << "=======================================" << endl;

	cout << "CPU time -  " << time.count() * 100<< endl;
	cout << "Basic GPU time - " << flatMatrixBasic(h_src, d_dst_for_basic, m, n_new) / 100 << endl;
	cout << "Shared memory GPU -  " << flatMatrixShared(h_src, d_dst_for_shared, m, n_new) / 100 << endl;
	
	cout << "=======================================" << endl;

		if (compare(d_dst, d_dst_for_shared, n_new * m)) {

			cout << "Done correctly\n\n\n" << endl;

			
			
		}else {
		cout << "\n\n\n" << endl;
		}

	print(d_dst_for_shared, m * 2, n_new / 2);
	cout << "=======================================" << endl;
	
	free(h_src);
	free(d_dst);
	free(d_dst_for_basic);
	free(d_dst_for_shared);

	return 0;
}