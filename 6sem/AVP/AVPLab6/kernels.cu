#include "hip/hip_runtime.h"
#include "kernels.cuh"


__global__ void kernel(
	unsigned char* input,
	unsigned char* output,
	int width,
	int dimY,
	int dimX
) {

	__shared__ uint32_t* iInput;

	const int xIdx = blockIdx.x * BLOCK_DIM_X + threadIdx.x;
	const int yIdx = blockIdx.y * BLOCK_DIM_Y + threadIdx.y;

	if (xIdx >= width || yIdx >= dimY)
	{
		return;
	}

	iInput = (uint32_t*)input;
	uint32_t q = 0, res = 0, offset = 0;
	int absoluteX, absoluteY, intPos = 0;
	for (int byteIdx = 0; byteIdx < 4; byteIdx++)
	{
		int positionBytes = xIdx * 4 + byteIdx;
		unsigned char subres = 0;
		for (int j = 0; j < 3; j++)
		{
			absoluteY = yIdx + j - 1;
			if (absoluteY < 0 || absoluteY >= dimY)
			{
				absoluteY = yIdx;
			}

			for (int i = 0; i < 3; i++)
			{
				absoluteX = positionBytes + i - 1;
				if (absoluteX < 0 || absoluteX >= dimX)
				{
					absoluteX = positionBytes;
				}
				offset = absoluteX / 4;
				intPos = absoluteX % 4;
				q = iInput[offset + absoluteY * width];
				subres += ((unsigned char*)&q)[intPos] * HF_H1_DEVICE_KERNEL[j][i];
			}
		}
		((unsigned char*)&res)[byteIdx] = subres;
	}
	__syncthreads();
	((uint32_t*)output)[xIdx + yIdx * width] = res;

}

__global__ void getChannelsGPU(
	unsigned char* sourceImageData,
	int dimX, // px
	 int dimY, // px
	unsigned char* rs,
	unsigned char* gs,
	unsigned char* bs
) {

	 const int xIndex = blockIdx.x * BLOCK_DIM_X + threadIdx.x; // reference in px
	 const int yIndex = blockIdx.y * BLOCK_DIM_Y + threadIdx.y; // line # in px
	if (xIndex >= dimX * 3 || yIndex >= dimY)
	{
		return;
	}
	int colorOffset = 0;
	rs[xIndex + yIndex * dimX] = sourceImageData[xIndex * 3 + colorOffset + yIndex * dimX * 3];
	colorOffset++;
	gs[xIndex + yIndex * dimX] = sourceImageData[xIndex * 3 + colorOffset + yIndex * dimX * 3];
	colorOffset++;
	bs[xIndex + yIndex * dimX] = sourceImageData[xIndex * 3 + colorOffset + yIndex * dimX * 3];
}

__global__ void compileChannelsGPU(
	unsigned char* r,
	unsigned char* g,
	unsigned char* b,
	int dimX,
	int dimY,
	unsigned char* output
) {

	const int xIndex = blockIdx.x * BLOCK_DIM_X + threadIdx.x; // reference in px
	const int yIndex = blockIdx.y * BLOCK_DIM_Y + threadIdx.y; // line # in px
	if (xIndex >= dimX * 3 || yIndex >= dimY)
	{
		return;
	}

	int colorOffset;
	int colorOffset1;
	colorOffset1 = 0;
	colorOffset = colorOffset1;

	output[xIndex * 3 + colorOffset + yIndex * dimX * 3] = r[xIndex + yIndex * dimX];
	colorOffset++;
	output[xIndex * 3 + colorOffset + yIndex * dimX * 3] = g[xIndex + yIndex * dimX];
	colorOffset++;
	output[xIndex * 3 + colorOffset + yIndex * dimX * 3] = b[xIndex + yIndex * dimX];
	colorOffset++;
}