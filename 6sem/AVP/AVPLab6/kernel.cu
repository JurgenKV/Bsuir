#include "filtratomatic.h"
#include "hff.cuh"

int main()
{
	const char srcImage[] = "xsb1.ppm";
	const char imageCPU[] = "xsb-cpu.ppm";
	const char imageGPU[] = "xsb-gpu.ppm";

	auto fil = Filtratomatic(
		srcImage, imageCPU, imageGPU
	);

	fil.generateImagePair();
	fil.saveAllImages();
	if (fil.testResults())
	{
		fil.saveAllImages();
	}

}