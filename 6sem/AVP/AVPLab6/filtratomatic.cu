#include "hip/hip_runtime.h"
#include "filtratomatic.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <windows.h>
#include <math.h>
#include <stdlib.h>
#include <stdint.h>
#include <iomanip>

#include "helper_image.h"
#include "kernels.cuh"

void 
initSourceImage (
	unsigned char* sourceImageData,
	unsigned char* r,
	unsigned char* g,
	unsigned char* b,
	int dimX, 
	int dimY
) {
	int dataIdx = 0;
	int pxIdx = 0;
	while (pxIdx < dimX * dimY)
	{
		r[pxIdx] = sourceImageData[dataIdx++];

		g[pxIdx] = sourceImageData[dataIdx++];

		b[pxIdx] = sourceImageData[dataIdx++];
		pxIdx++;

	}
}

unsigned char* 
compileChannels (
	unsigned char* r, 
	unsigned char* g, 
	unsigned char* b, 
	int dimX, 
	int dimY
) {
	auto* image = new unsigned char[dimX * dimY * 3];

	int dataIdx = 0;
	int pxIdx = 0;
	while (pxIdx < dimX * dimY)
	{
		image[dataIdx++] = r[pxIdx];
		image[dataIdx++] = g[pxIdx];
		image[dataIdx++] = b[pxIdx];
		pxIdx++;
	}
	return image;
}

Filtratomatic::Filtratomatic (
	const char* source,
	const char* cpuPath,
	const char* gpuPath
) : sourcePath(source), cpuPath(cpuPath), gpuPath(gpuPath)
{
	 dimX = 0;
	 dimY = 0;
	channels = 0;
	 unsigned char* sourceImageData = NULL;
	__loadPPM(sourcePath, &sourceImageData, &dimX, &dimY, &channels);
	//std::cout << channels << std::endl;
	padding = 4 - (dimX % 4);
	paddedX = dimX + padding;
	auto* paddedSourceImageData = new unsigned char[paddedX * dimY * 3];

	for (int i = 0; i < dimY; i++)
	{
		for (int j = 0; j < dimX; j++)
		{
			for (int colorIdx = 0; colorIdx < channels; colorIdx++)
			{
				paddedSourceImageData[(j + i * paddedX) * channels + colorIdx] = 
					sourceImageData[(j + i * dimX) * channels + colorIdx];
			}
		}
	}
	dimX = paddedX;

	sourceR = new unsigned char[dimX * dimY];
	sourceG = new unsigned char[dimX * dimY];
	sourceB = new unsigned char[dimX * dimY];
	/*int k = 0;
	while (k < paddedX * dimY * 3 ) {
		std::cout << paddedSourceImageData[k];
		k++;
	}*/

	initSourceImage(paddedSourceImageData, sourceR, sourceG, sourceB, dimX, dimY);


	sourceImageRawFreakingDataBecauseSomeoneJustHatesAllOfHumanity = paddedSourceImageData;
}

bool 
Filtratomatic::testResults()
{
	for (int i = 0; i < dimY * dimX * 3; i++)
	{
		if (abs(reference[i] - gpuImage[i]) > 1) {
			std::cerr << "[ERR] gpu result does not match reference, delta " << reference[i] - gpuImage[i] << std::endl;
			return false;
		}
	}
	std::cerr << "[AOK] gpu result matches reference" << std::endl;
	return true;
}


MeasuredResult*
generateReferenceChannel (
	int dimX, 
	int dimY, 
	unsigned char* source
) {
	std::chrono::high_resolution_clock::time_point tpStart;
	std::chrono::high_resolution_clock::time_point tpEnd;
	auto* result = new MeasuredResult;
	auto* reference = new unsigned char[dimX * dimY];
	int refPosition = 0;

	if (reference == nullptr)
	{
		throw std::runtime_error("[ERR] unable to allocate reference memory");
	}

	tpStart = std::chrono::high_resolution_clock::now();
	for (int y = 0; y < dimY; y++)
	{
		for (int x = 0; x < dimX; x++)
		{
			unsigned char result = 0;
			for (int j = 0; j < 3; j++)
			{
				for (int i = 0; i < 3; i++)
				{
					int adjustedX = x + i - 1;
					if (adjustedX < 0)
					{
						adjustedX = 0;
					}
					if (adjustedX == dimX)
					{
						adjustedX = dimX - 1;
					}

					int adjustedY = y + j - 1;
					if (adjustedY < 0)
					{
						adjustedY = 0;
					}
					if (adjustedY == dimY)
					{
						adjustedY = dimY - 1;
					}

					refPosition = dimX * adjustedY + adjustedX;
					result += source[refPosition] * HF_H1_KERNEL[j][i];
				}
			}
			reference[y * dimX + x] = result;
		}
	}
	tpEnd = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> diff = tpEnd - tpStart;

	result->ptr = reference;
	result->elapsedTime = diff.count();

	return result;
}

void 
Filtratomatic::generateReference()
{
	auto* rResult = generateReferenceChannel(dimX, dimY, sourceR);
	auto* gResult = generateReferenceChannel(dimX, dimY, sourceG);
	auto* bResult = generateReferenceChannel(dimX, dimY, sourceB);

	reference = compileChannels(
		rResult->ptr,
		gResult->ptr,
		bResult->ptr,
		dimX, dimY
	);

	float combinedTime = rResult->elapsedTime + gResult->elapsedTime + bResult->elapsedTime;
	std::cout
		<< "[INF] reference generated in "
		<< combinedTime
		<< "s " << std::endl;
}

MeasuredResult* 
generateGPUChannels (
	int dimX, 
	int dimY, 
	unsigned char* rawSourceImageData
) {
	auto* result = new MeasuredResult;
	 hipStream_t streamR, streamG, streamB;
	 hipStreamCreate(&streamR);
	 hipStreamCreate(&streamG);
	 hipStreamCreate(&streamB);

	long long int imgSz = dimX * dimY;
	hipEvent_t ceStart, ceEnd;
	unsigned char* gpuResultR, *gpuResultG, *gpuResultB;
	unsigned char* gpuSourceR, *gpuSourceG, *gpuSourceB, *gpuSourceImage;

	hipMalloc((void**)&gpuResultR, imgSz);
	hipMalloc((void**)&gpuResultG, imgSz);
	hipMalloc((void**)&gpuResultB, imgSz);
	hipMalloc((void**)&gpuSourceR, imgSz);
	hipMalloc((void**)&gpuSourceG, imgSz);
	hipMalloc((void**)&gpuSourceB, imgSz);
	hipMalloc((void**)&gpuSourceImage, imgSz*3);

	hipMemcpy(gpuSourceImage, rawSourceImageData, imgSz * 3, hipMemcpyHostToDevice);

	dim3 dimGridColorSep(
		(int)ceil((double)dimX / (double)BLOCK_DIM_X),
		(int)ceil((double)dimY / (double)BLOCK_DIM_Y)
	);
	dim3 dimBlockColorSep(BLOCK_DIM_X, BLOCK_DIM_Y);

	int width = dimX / 4;
	dim3 dimGrid(
		(int)ceil((double)width / (double)BLOCK_DIM_X),
		(int)ceil((double)dimY / (double)BLOCK_DIM_Y)
	);
	dim3 dimBlock(BLOCK_DIM_X, BLOCK_DIM_Y);

	dim3 dimGridColorJoin(
		(int)ceil((double)dimX / (double)BLOCK_DIM_X),
		(int)ceil((double)dimY / (double)BLOCK_DIM_Y)
	);
	dim3 dimBlockColorJoin(BLOCK_DIM_X, BLOCK_DIM_Y);

	auto* gpuImageResult = new unsigned char[imgSz * 3];
	hipMalloc((void**)&gpuImageResult, imgSz * 3);

	hipEventCreate(&ceStart);
	hipEventCreate(&ceEnd);
	hipEventRecord(ceStart, nullptr);

	getChannelsGPU <<< dimGridColorSep, dimBlockColorSep >>> (gpuSourceImage, dimX, dimY, gpuSourceR, gpuSourceG, gpuSourceB);
	kernel <<< dimGrid, dimBlock, 0, streamR >>> (gpuSourceR, gpuResultR, width, dimY, dimX);
	kernel <<< dimGrid, dimBlock, 0, streamG >>> (gpuSourceG, gpuResultG, width, dimY, dimX);
	kernel <<< dimGrid, dimBlock, 0, streamB >>> (gpuSourceB, gpuResultB, width, dimY, dimX);	
	compileChannelsGPU <<< dimGridColorJoin, dimBlockColorJoin >>> (gpuResultR, gpuResultG, gpuResultB, dimX, dimY, gpuImageResult);
	
	hipEventRecord(ceEnd, nullptr);
	hipEventSynchronize(ceEnd);

	float duration;
	duration = 0;
	hipEventElapsedTime(&duration, ceStart, ceEnd);

	unsigned char* res = new unsigned char[imgSz * 3];
	hipMemcpy(res, gpuImageResult, imgSz * 3, hipMemcpyDeviceToHost);

	result->elapsedTime = duration;
	result->ptr = res;

	hipFree(gpuResultR);
	hipFree(gpuResultG);
	hipFree(gpuResultB);

	hipFree(gpuSourceR);
	hipFree(gpuSourceG);
	hipFree(gpuSourceB);

	hipDeviceReset();

	return result;
}


void 
Filtratomatic::generateGPU () 
{
	auto* result = generateGPUChannels(dimX, dimY, sourceImageRawFreakingDataBecauseSomeoneJustHatesAllOfHumanity);
	gpuImage = result->ptr;

	std::cout
		<< "[INF] gpu image generated in "
		<< result->elapsedTime / 1000
		<< "s " << std::endl;
}

void 
Filtratomatic::saveAllImages()
{
	dimX -= padding;
	auto* unpaddedRef = new unsigned char[dimX * dimY * 3];
	auto* unpaddedGPU = new unsigned char[dimX * dimY * 3];

	for (int i = 0; i < dimY; i++)
	{
		for (int j = 0; j < dimX; j++)
		{
			for (int colorIdx = 0; colorIdx < channels; colorIdx++)
			{
				unpaddedRef[(j + i * dimX) * channels + colorIdx] =
					reference[(j + i * paddedX) * channels + colorIdx];
				unpaddedGPU[(j + i * dimX) * channels + colorIdx] =
					gpuImage[(j + i * paddedX) * channels + colorIdx];
			}
		}
	}

	  __savePPM(cpuPath, unpaddedRef, dimX, dimY, channels);
	  __savePPM(gpuPath, unpaddedGPU, dimX, dimY, channels);
}


void 
Filtratomatic::generateImagePair () 
{
	std::chrono::high_resolution_clock::time_point tpStart;
	std::chrono::high_resolution_clock::time_point tpEnd;
	tpStart = std::chrono::high_resolution_clock::now();
	generateReference();
	generateGPU();
	tpEnd = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> diff = tpEnd - tpStart;
	std::cout
		<< "[INF] total time "
		<< diff.count()
		<< "s " << std::endl;
}

